#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "cuda_device_launch_parameters.h"
float dt=0.01;

__global__ void Compute(int *p,int *v,int* m,int *N){
	int id=threadIdx.x,i;
	double Force=0;
	double G=6.674*pow(10,-11);
	for(i=0;i<*N;i++){
		if(i==id)
			continue;
		k=p[id]-p[i];
		Force+=(-1)*G*m[id]*m[i]*(k)/fabs(pow(k,3));
	}
	double a=Force/m[id];
	double newV=v[id]+a*dt;
	double newP=v[id]*t+(1/2)*a*pow(dt,2);
	v[id]=newV;
	p[id]=newP;
}

int main(void){
	int N,i;
	float T;
	printf("Enter number of objects:");
	scanf("%d",&N);
	printf("Enter evaluation time:");
	scanf("%d",&T);
	int steps=int(T/dt);
	double p[N],v[N],m[N];

	printf("Enter the initial position of %d objects:\n",N);
	for(i=0;i<N;i++)
		scanf("%lf",&p[i]);
	printf("Enter the initial velocity of %d objects:\n",N);
	for(i=0;i<N;i++)
		scanf("%lf",&v[i]);
	printf("Enter the mass of %d objects:\n",N);
	for(i=0;i<N;i++)
		scanf("%lf",&m[i]);

	double *d_p,*d_v,*d_m,*d_n;
	int size=N*sizeof(double);

	hipMalloc((void**)&d_p,size);
	hipMalloc((void**)&d_v,size);
	hipMalloc((void**)&d_m,size);
	hipMalloc((void**)&d_n,sizeof(int));

	hipMemcpy(d_p,&p,size,hipMemcpyHostToDevice);
	hipMemcpy(d_v,&v,size,hipMemcpyHostToDevice);
	hipMemcpy(d_m,&m,size,hipMemcpyHostToDevice);
	hipMemcpy(d_n,&N,sizeof(int),hipMemcpyHostToDevice);

	for(i=0;i<steps;i++){
		Compute<<<1,N>>>(d_p,d_v,d_m,d_n);
	}

	hipMemcpy(&p,d_p,size,hipMemcpyDeviceToHost);
	hipMemcpy(&v,d_v,size,hipMemcpyDeviceToHost);

	printf("\tPosition\tVeocity");
	for(i=0;i<N;i++){
		printf("Object %d:\t%lf\t%lf\n",i+1,p[i],v[i]);
	}

	return 0;
}
